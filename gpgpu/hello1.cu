

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
   GPU用strcpy
   GPUの関数を指定するには__device__をつける
 **/
__device__ void dev_strcpy(char *dst, const char *src) {
    while (*dst++ = *src++);
}

/**
   GPU側エントリ		
**/
__global__ void gen_hello(char *A) {
    dev_strcpy(A, "Hello, World");
}

int main() {
    char *d_hello;

    // ホストでメモリを確保
    char hello[128];

    // GPU側のメモリ（デバイスメモリ）を確保
    hipMalloc((void**)&d_hello, 128);

    // gen_hello呼び出し
    gen_hello<<<1,1>>>(d_hello);

    // GPU側データの取得
    hipMemcpy(hello, d_hello, 128, hipMemcpyDeviceToHost);

    // 確保したメモリを解放
    hipFree(d_hello);

    // 出力
    puts(hello);
}
